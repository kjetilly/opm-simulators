#include "hip/hip_runtime.h"
/*
  Copyright 2025 Equinor ASA

  This file is part of the Open Porous Media project (OPM).
  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <boost/test/tools/old/interface.hpp>
#include <config.h>
#include <stdexcept>

#define BOOST_TEST_MODULE TestThrowMacrosOnGPU

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <boost/test/unit_test.hpp>
#include <opm/common/ErrorMacros.hpp>
#include <opm/simulators/linalg/gpuistl/detail/gpu_safe_call.hpp>

namespace {
__global__ void codeThatContainsMacros(bool call) {
    if (call) {
        OPM_THROW(std::logic_error, "Something went wrong");
        OPM_THROW_NOLOG(std::logic_error, "Something went wrong");
        OPM_THROW_PROBLEM(std::logic_error, "Something went wrong");
    }
    OPM_ERROR_IF(!call, "Something went horribly wrong");
}

// TODO: Check if this is better on HIP
#if 0 // I am leaving this here to show that this is not possible due to limitations in CUDA
      // the assert will indeed cause an error, but the CUDA context will be broken for
      // the rest of the lifetime of the process, see 
      // https://forums.developer.nvidia.com/t/how-to-clear-cuda-errors/296393/5
__global__ void checkThrow() {
    OPM_THROW(std::logic_error, "Something went wrong");
}

__global__ void checkThrowNoLog() {
    OPM_THROW_NOLOG(std::logic_error, "Something went wrong");
}

__global__ void checkThrowProblem() {
    OPM_THROW_PROBLEM(std::logic_error, "Something went wrong");
}

__global__ void checkErrorIf() {
    OPM_ERROR_IF(true, "Something went horribly wrong");
}
#endif
}

BOOST_AUTO_TEST_CASE(TestKernel)
{
    OPM_GPU_SAFE_CALL(hipDeviceSynchronize());
    OPM_GPU_SAFE_CALL(hipGetLastError());
    codeThatContainsMacros<<<1, 1>>>(false);
    OPM_GPU_SAFE_CALL(hipDeviceSynchronize());
    OPM_GPU_SAFE_CALL(hipGetLastError());

    #if 0 // I am leaving this here to show that this is not possible due to limitations in CUDA
          // the assert will indeed cause an error, but the CUDA context will be broken for
          // the rest of the lifetime of the process, see 
          // https://forums.developer.nvidia.com/t/how-to-clear-cuda-errors/296393/5
    codeThatContainsMacros<<<1, 1>>>(true);
    // Make sure this actually throws
    BOOST_CHECK_THROW(OPM_GPU_SAFE_CALL(hipDeviceSynchronize()), std::runtime_error);
    OPM_GPU_SAFE_CALL(hipDeviceReset());
    OPM_GPU_SAFE_CALL(hipGetLastError());

    checkThrow<<<1, 1>>>();
    BOOST_CHECK_THROW(OPM_GPU_SAFE_CALL(hipDeviceSynchronize()), std::runtime_error);
    OPM_GPU_SAFE_CALL(hipDeviceReset());
    OPM_GPU_SAFE_CALL(hipGetLastError());

    checkThrowNoLog<<<1, 1>>>();
    BOOST_CHECK_THROW(OPM_GPU_SAFE_CALL(hipDeviceSynchronize()), std::runtime_error);
    OPM_GPU_SAFE_CALL(hipDeviceReset());
    OPM_GPU_SAFE_CALL(hipGetLastError());

    checkThrowProblem<<<1, 1>>>();
    BOOST_CHECK_THROW(OPM_GPU_SAFE_CALL(hipDeviceSynchronize()), std::runtime_error);
    OPM_GPU_SAFE_CALL(hipDeviceReset());
    OPM_GPU_SAFE_CALL(hipGetLastError());

    checkErrorIf<<<1, 1>>>();
    BOOST_CHECK_THROW(OPM_GPU_SAFE_CALL(hipDeviceSynchronize()), std::runtime_error);
    OPM_GPU_SAFE_CALL(hipDeviceReset());
    OPM_GPU_SAFE_CALL(hipGetLastError());
    #endif
}

BOOST_AUTO_TEST_CASE(TestOutsideKernel) 
{
    // This is to make sure that the macros work outside of kernels but inside a .cu file
    // ie. inside a file compiled by nvcc/hipcc.
    BOOST_CHECK_THROW(OPM_THROW(std::runtime_error, "THROW"), std::runtime_error);
    BOOST_CHECK_THROW(OPM_THROW_NOLOG(std::runtime_error, "THROW_NOLOG"), std::runtime_error);
    BOOST_CHECK_THROW(OPM_THROW_PROBLEM(std::runtime_error, "THROW_PROBLEM"), std::runtime_error);
    BOOST_CHECK_THROW(OPM_ERROR_IF(true, "ERROR_IF"), std::logic_error);
}
