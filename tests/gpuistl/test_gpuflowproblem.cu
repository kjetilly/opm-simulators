#include "hip/hip_runtime.h"
/*
  Copyright 2025 SINTEF AS
  This file is part of the Open Porous Media project (OPM).
  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
// #undef NDEBUG
#include <config.h>
// #undef NDEBUG

#include <string>

#define BOOST_TEST_MODULE TestFlowProblemGpu

#include <boost/test/unit_test.hpp>
#include <opm/material/densead/Evaluation.hpp>
#include <opm/material/fluidmatrixinteractions/EclMaterialLawManagerSimple.hpp>
#include <opm/simulators/linalg/gpuistl/DualBuffer.hpp>
#include <opm/simulators/linalg/gpuistl/GpuBuffer.hpp>

#include <opm/models/blackoil/blackoilmodel.hh>
#include <opm/models/discretization/common/tpfalinearizer.hh>
#include <opm/models/utils/simulator.hh>

#include <opm/simulators/flow/FlowProblemBlackoil.hpp>
#include <opm/simulators/flow/FlowProblemBlackoilGpu.hpp>
#include <opm/simulators/flow/FlowProblemBlackoilProperties.hpp>
#include <opm/simulators/utils/moduleVersion.hpp>

#include <opm/simulators/flow/BlackoilModelParameters.hpp>
#include <opm/simulators/flow/FlowGenericVanguard.hpp>
#include <opm/simulators/flow/FlowProblemBlackoil.hpp>
#include <opm/simulators/flow/FlowProblemBlackoilProperties.hpp>
#include <opm/simulators/flow/equil/EquilibrationHelpers.hpp>
#include <opm/simulators/linalg/gpuistl/DualBuffer.hpp>
#include <opm/simulators/linalg/gpuistl/GpuBuffer.hpp>
#include <opm/simulators/linalg/gpuistl/GpuView.hpp>
#include <opm/simulators/linalg/gpuistl/detail/gpu_safe_call.hpp>
#include <opm/simulators/linalg/gpuistl/gpu_smart_pointer.hpp>
#include <opm/simulators/linalg/parallelbicgstabbackend.hh>
#include <opm/simulators/wells/BlackoilWellModel.hpp>

#include <utility>

#include <hip/hip_runtime.h>

/*
Functionality requested for the blackoil flow problem on gpu:
[X] indicates that the functionality is added and verified with unit test
[-] indicates that the functionality does not seem to be used in spe11
[ ] indicates that the functionality is not added yet

[X] - problem.model().linearizer().getLinearizationType()
[X] - problem.satnumRegionIndex(globalSpaceIdx)
[X] - problem.materialLawParams(globalSpaceIdx)
[X] - problem.rockCompressibility(globalSpaceIdx)
[X] - problem.rockReferencePressure(globalSpaceIdx)
[X] - problem.porosity(globalSpaceIdx, timeIdx)
[-] - problem.maxOilVaporizationFactor(timeIdx, globalSpaceIdx)
[-] - problem.maxGasDissolutionFactor(timeIdx, globalSpaceIdx)
[-] - problem.maxOilSaturation(globalSpaceIdx)
[-] - problem.template rockCompPoroMultiplier<Evaluation>(*this, globalSpaceIdx)
[X] - problem.updateRelperms(mobility_, dirMob_, fluidState_, globalSpaceIdx)
[X] - problem.template rockCompTransMultiplier<Evaluation>(*this, globalSpaceIdx)

*/

#include <opm/material/fluidstates/BlackOilFluidState.hpp>
#include <opm/material/fluidsystems/BlackOilFluidSystem.hpp>
#include <opm/material/fluidsystems/BlackOilFluidSystemNonStatic.hpp>

#include <opm/input/eclipse/Deck/Deck.hpp>
#include <opm/input/eclipse/EclipseState/EclipseState.hpp>
#include <opm/input/eclipse/Parser/Parser.hpp>
#include <opm/input/eclipse/Python/Python.hpp>
#include <opm/input/eclipse/Schedule/Schedule.hpp>
#include <tests/common_type_tag.hpp>
// #include <tests/load_data.hpp>
// #include <tests/load_data.cpp>

#include <iostream>
#include <memory>
#include <type_traits>
// #include <dune/common/mpihelper.hh>
#include <dune/common/parallel/mpihelper.hh>
#include <opm/models/utils/start.hh>

#include <opm/simulators/linalg/gpuistl/gpu_smart_pointer.hpp>

template <class ProblemView>
__global__ void
satnumFromFlowProblemBlackoilGpu(ProblemView prob, unsigned short* res)
{
    *res = prob.satnumRegionIndex(0);
}

template <class ProblemView>
__global__ void
linTypeFromFlowProblemBlackoilGpu(ProblemView prob, Opm::LinearizationType* res)
{
    *res = prob.model().linearizer().getLinearizationType();
}

template <class ProblemView>
__global__ void
rockCompressibilityFromFlowProblemBlackoilGpu(ProblemView prob, double* res)
{
    *res = prob.rockCompressibility(0);
}

template <class ProblemView>
__global__ void
porosityFromFlowProblemBlackoilGpu(ProblemView prob, double* res)
{
    *res = prob.porosity(0, 0);
}

template <class ProblemView>
__global__ void
rockReferencePressureFromFlowProblemBlackoilGpu(ProblemView prob, double* res)
{
    *res = prob.rockReferencePressure(0);
}

template <class ProblemView>
__global__ void
materialLawParamsCallable(ProblemView prob)
{
    auto matLawParams = prob.materialLawParams(0);
}

template <class DirMobPtr, class ProblemView, class MobArr, class FluidState>
__global__ void
updateRelPermsFromFlowProblemBlackoilGpu(ProblemView prob, MobArr mob, FluidState fs)
{
    auto dirPtr = DirMobPtr(); // produces nullptr, this value is not used in the function, but should match signature
    prob.updateRelperms(mob, dirPtr, fs, 0);
}


BOOST_AUTO_TEST_CASE(TestInstantiateGpuFlowProblem)
{
    std::cout << __LINE__ << std::endl;
    using TypeTag = Opm::Properties::TTag::FlowSimpleProblem;
    std::cout << __LINE__ << std::endl;
    // TODO: will this actually refer to the very_simple_deck.DATA inside the gpuistl folder,
    // TODO: do we need to keep track of the path since it can be hipified?
    const std::string filename = "very_simple_deck.DATA";
        std::cout << __LINE__ << std::endl;

    std::vector<std::string> args = {filename, "very_simple_deck.DATA", ""};
        std::cout << __LINE__ << std::endl;

    std::vector<char*> argv2;
        std::cout << __LINE__ << std::endl;

    for (auto& arg : args) {
            std::cout << __LINE__ << std::endl;

        argv2.push_back(static_cast<char*>(arg.data()));
            std::cout << __LINE__ << std::endl;

    }
        std::cout << __LINE__ << std::endl;

    using TypeTag = Opm::Properties::TTag::FlowSimpleProblem;
        std::cout << __LINE__ << std::endl;

    // auto mainObject = Opm::Main(argv2.size() - 1, argv2.data());
    auto mainObject = Opm::Main(filename, false, false);
        std::cout << __LINE__ << std::endl;

    // mainObject.runStatic<TypeTag>();
    auto mainFlow = mainObject.gimmeFlowMain<TypeTag>();
        std::cout << __LINE__ << std::endl;

    std::cout << "Got mainFlow" << std::endl;
    mainFlow->execute();
    std::cout << "Executed mainFlow" << std::endl;
    auto simulator = mainFlow->getSimulator();
    std::cout << "Got simulator" << std::endl;
    auto& problem = simulator->problem();
    std::cout << "Got problem" << std::endl;
    #if 0
    auto problemGpuBuf
        = Opm::gpuistl::copy_to_gpu<double, Opm::gpuistl::GpuBuffer, Opm::gpuistl::DualBuffer, TypeTag, TypeTag>(
            problem);

    fmt::println("From callback");
    // auto problemGpuBuf = Opm::gpuistl::
    //     copy_to_gpu<double, Opm::gpuistl::GpuBuffer, Opm::gpuistl::DualBuffer, TypeTag, TypeTag>(problem);
    fmt::println("Copied to GPU");
    auto problemGpuView = Opm::gpuistl::make_view<Opm::gpuistl::GpuView, Opm::gpuistl::ValueAsPointer>(problemGpuBuf);

    fmt::println("At line {}", __LINE__);

    auto satNumOnGpu = Opm::gpuistl::make_gpu_unique_ptr<unsigned short>(0);
    satnumFromFlowProblemBlackoilGpu<<<1, 1>>>(problemGpuView, satNumOnGpu.get());
    const auto satNumOnCpu = Opm::gpuistl::copyFromGPU(satNumOnGpu);
    BOOST_CHECK_EQUAL(satNumOnCpu, problem.satnumRegionIndex(0));
    fmt::println("At line {}", __LINE__);

    auto linTypeOnGpu = Opm::gpuistl::make_gpu_unique_ptr<Opm::LinearizationType>(Opm::LinearizationType{});
    linTypeFromFlowProblemBlackoilGpu<<<1, 1>>>(problemGpuView, linTypeOnGpu.get());
    const auto linTypeOnCpu = Opm::gpuistl::copyFromGPU(linTypeOnGpu);
    auto linTypeFromCPUSimulator = problem.model().linearizer().getLinearizationType();
    BOOST_CHECK_EQUAL(linTypeOnCpu.type, linTypeFromCPUSimulator.type);
    fmt::println("At line {}", __LINE__);

    auto rockCompressibilityOnGpu = Opm::gpuistl::make_gpu_unique_ptr<double>(0.0);
    rockCompressibilityFromFlowProblemBlackoilGpu<<<1, 1>>>(problemGpuView, rockCompressibilityOnGpu.get());
   
    const auto rocmCompressibilityOnCpu = Opm::gpuistl::copyFromGPU(rockCompressibilityOnGpu);
    BOOST_CHECK_EQUAL(rocmCompressibilityOnCpu, problem.rockCompressibility(0));
    fmt::println("At line {}", __LINE__);

    auto  porosityOnGpu = Opm::gpuistl::make_gpu_unique_ptr<double>(0.0);
    porosityFromFlowProblemBlackoilGpu<<<1, 1>>>(problemGpuView, porosityOnGpu.get());
    const auto porosityOnCpu = Opm::gpuistl::copyFromGPU(porosityOnGpu);
    BOOST_CHECK_EQUAL(porosityOnCpu, problem.porosity(0, 0));

    auto referencePressureOnGpu = Opm::gpuistl::make_gpu_unique_ptr<double>(0.0);
    rockReferencePressureFromFlowProblemBlackoilGpu<<<1, 1>>>(problemGpuView, referencePressureOnGpu.get());
    fmt::println("At line {}", __LINE__);
    const auto referencePressureOnCpu = Opm::gpuistl::copyFromGPU(referencePressureOnGpu);
    BOOST_CHECK_EQUAL(referencePressureOnCpu, problem.rockReferencePressure(0));
    
    materialLawParamsCallable<<<1, 1>>>(problemGpuView);
    fmt::println("At line {}", __LINE__);

    using FluidSystem = Opm::BlackOilFluidSystem<double>;
    using Evaluation = Opm::DenseAd::Evaluation<double, 2>;
    using Scalar = double;
    // using DirectionalMobilityPtr = Utility::CopyablePtr<DirectionalMobility<TypeTag, Evaluation>>;
    using DirectionalMobilityPtr = Opm::Utility::CopyablePtr<Opm::DirectionalMobility<TypeTag>>;
    fmt::println("At line {}", __LINE__);


    // Create the fluid system
    std::string deckString1;
    {
        std::ifstream deckFile(filename);
        if (!deckFile) {
            throw std::runtime_error("Failed to open deck file: " + filename);
        }
        std::stringstream buffer;
        buffer << deckFile.rdbuf();
        deckString1 = buffer.str();
    }
    Opm::Parser parser;
    auto deck = parser.parseString(deckString1);
    auto python = std::make_shared<Opm::Python>();
    Opm::EclipseState eclState(deck);
    Opm::Schedule schedule(deck, eclState, python);
    fmt::println("At line {}", __LINE__);

    FluidSystem::initFromState(eclState, schedule);
    auto& dynamicFluidSystem = FluidSystem::getNonStaticInstance();
    auto dynamicGpuFluidSystemBuffer
        = ::Opm::gpuistl::copy_to_gpu<::Opm::gpuistl::GpuBuffer, double>(dynamicFluidSystem);
    auto dynamicGpuFluidSystemView = ::Opm::gpuistl::make_view<::Opm::gpuistl::GpuView, ::Opm::gpuistl::ValueAsPointer>(
        dynamicGpuFluidSystemBuffer);
    auto gpufluidstate
        = Opm::BlackOilFluidState<double, decltype(dynamicGpuFluidSystemView)>(dynamicGpuFluidSystemView);
    // Create MobArr
    double testValue = 0.5;
    // Create an array of Evaluations on CPU
    using MobArr = std::array<Evaluation, 2>;
    MobArr cpuMobArray;
    cpuMobArray[0] = Evaluation(testValue, 0);
    cpuMobArray[1] = Evaluation(testValue, 1);
    fmt::println("At line {}", __LINE__);
    fmt::println("At line {}", __LINE__);

    updateRelPermsFromFlowProblemBlackoilGpu<DirectionalMobilityPtr>
         <<<1, 1>>>(problemGpuView, cpuMobArray, gpufluidstate);
    fmt::println("At line {}", __LINE__);

    OPM_GPU_SAFE_CALL(hipDeviceSynchronize());
    fmt::println("At line {}", __LINE__);
    #endif
}
